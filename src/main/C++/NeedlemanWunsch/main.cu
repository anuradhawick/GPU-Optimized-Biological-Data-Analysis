#include "hip/hip_runtime.h"
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <vector>
#include <stack>

#include "../commons/matrixUtils.h"

using namespace std;

void executeCPU(string X, string Y);
void getAlignments(string X, string Y, Matrix matrix);
void executeGPU(string X, string Y);
__global__ void needlemanKernel(Matrix mat, string X, string Y);

typedef struct {
	char *content;
	int length;
} CUDAstring;

int GAP = -1, MISMATCH = -1, MATCH = 1;

int main() {
	// Size of vectors
	string X("GGTTGACTA");
	string Y("TGTTACGG");
	struct timeval start, end;

	gettimeofday(&start, NULL);
	executeCPU(X, Y);
	gettimeofday(&end, NULL);
	cout << "CPU calculation ended in "
			<< (end.tv_sec - start.tv_sec) * 1000
					+ (end.tv_usec - start.tv_usec) / 1000 << endl;

	gettimeofday(&start, NULL);
	executeGPU(X, Y);
	gettimeofday(&end, NULL);
	cout << "CPU calculation ended in "
			<< (end.tv_sec - start.tv_sec) * 1000
					+ (end.tv_usec - start.tv_usec) / 1000 << endl;

	return 0;
}

void executeCPU(string X, string Y) {
	int xs = X.size() + 1;
	int ys = Y.size() + 1;

	Matrix matrix = mallocMatrix(xs, ys);

	for (int i = 0; i < xs; i++) {
		matrix.elements[i] = GAP * i;
	}

	for (int j = 0; j < ys; j++) {
		matrix.elements[j * xs] = GAP * j;
	}

	for (int i = 1; i < ys; i++) {
		for (int j = 1; j < xs; j++) {
			int matchVal, yGapVal, xGapVal;
			// Match/mismatch
			if (Y[i - 1] == X[j - 1]) {
				matchVal = matrix.elements[(i - 1) * xs + j - 1] + MATCH;
			} else {
				matchVal = matrix.elements[(i - 1) * xs + j - 1] + MISMATCH;
			}
			// X Gap
			xGapVal = matrix.elements[(i - 1) * xs + j] + GAP;
			// Y Gap
			yGapVal = matrix.elements[(i) * xs + j - 1] + GAP;

			matrix.elements[i * xs + j] =
					(matchVal > xGapVal ? matchVal : xGapVal) > yGapVal ?
							(matchVal > xGapVal ? matchVal : xGapVal) : yGapVal;
		}
	}

	for (int i = 0; i < ys; i++) {
		for (int j = 0; j < xs; j++) {
			cout << matrix.elements[i * xs + j] << "\t";
		}
		cout << endl;
	}

	getAlignments(X, Y, matrix);
}

__global__ void needlemanKernel(Matrix matrix, CUDAstring X, CUDAstring Y,
		int MATCH, int MISMATCH, int GAP) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int xs = X.length + 1;
	int ys = Y.length + 1;

	if (i == 0 | j == 0 | i >= ys | j >= xs)
		return;

	int matchVal, yGapVal, xGapVal;
	// Match/mismatch
	if (Y.content[i - 1] == X.content[j - 1]) {
		matchVal = matrix.elements[(i - 1) * xs + j - 1] + MATCH;
	} else {
		matchVal = matrix.elements[(i - 1) * xs + j - 1] + MISMATCH;
	}
	// X Gap
	xGapVal = matrix.elements[(i - 1) * xs + j] + GAP;
	// Y Gap
	yGapVal = matrix.elements[(i) * xs + j - 1] + GAP;

	matrix.elements[i * xs + j] =
			(matchVal > xGapVal ? matchVal : xGapVal) > yGapVal ?
					(matchVal > xGapVal ? matchVal : xGapVal) : yGapVal;
}

void executeGPU(string X, string Y) {
	int xs = X.size() + 1;
	int ys = Y.size() + 1;
	size_t size = xs * ys * sizeof(int);

	Matrix matrix = mallocMatrix(xs, ys);
	Matrix d_matrix;
	d_matrix.width = matrix.width;
	d_matrix.height = matrix.height;

	CUDAstring d_X;
	d_X.length = X.size();
	char x_char[X.size()];
	strcpy(x_char, X.c_str());

	CUDAstring d_Y;
	d_Y.length = Y.size();
	char y_char[Y.size()];
	strcpy(y_char, Y.c_str());

	for (int i = 0; i < xs; i++) {
		matrix.elements[i] = GAP * i;
	}

	for (int j = 0; j < ys; j++) {
		matrix.elements[j * xs] = GAP * j;
	}

	hipMalloc(&d_matrix.elements, size);
	hipMalloc(&d_X.content, X.size() * sizeof(char));
	hipMalloc(&d_Y.content, Y.size() * sizeof(char));

	hipMemcpy(d_matrix.elements, matrix.elements, size,
			hipMemcpyHostToDevice);
	hipMemcpy(d_X.content, x_char, X.size() * sizeof(char),
			hipMemcpyHostToDevice);
	hipMemcpy(d_Y.content, y_char, Y.size() * sizeof(char),
			hipMemcpyHostToDevice);

	// Number of threads in each thread block
	dim3 threadsPerBlock(32, 32);

	// Number of thread blocks in grid
//	dim3 numBlocks((xs * ys + 32) / threadsPerBlock.x,
//			(xs * ys + 32) / threadsPerBlock.y);
	dim3 numBlocks(1, 1);

	for (int k = 0; k < 100; ++k) {
		needlemanKernel<<<numBlocks, threadsPerBlock>>>(d_matrix, d_X, d_Y,
				MATCH, MISMATCH, GAP);
	}

	hipDeviceSynchronize();
	hipMemcpy(matrix.elements, d_matrix.elements, size,
			hipMemcpyDeviceToHost);

	hipFree(d_matrix.elements);
	hipFree(d_X.content);
	hipFree(d_Y.content);

	for (int i = 0; i < ys; i++) {
		for (int j = 0; j < xs; j++) {
			cout << matrix.elements[i * xs + j] << "\t";
		}
		cout << endl;
	}

	getAlignments(X, Y, matrix);
}

void getAlignments(string X, string Y, Matrix matrix) {
	int xs = X.size() + 1;
	int ys = Y.size() + 1;

	int i = ys - 1;
	int j = xs - 1;
	bool hadPath = true;

	vector < stack<int*> > stList = vector<stack<int*> >();
	stack<int*> st = stack<int*>();

	int *arr = (int*) malloc(2 * sizeof(int));
	arr[0] = i;
	arr[1] = j;
	st.push(arr);
	stList.push_back(st);

	while (hadPath) {
		hadPath = false;
		vector < stack<int*> > temp = stList;
		stList = vector<stack<int*> >();

		for (int ii = 0; ii < temp.size(); ++ii) {
			int* pointer = temp[ii].top();
			i = pointer[0];
			j = pointer[1];

			if (i - 1 >= 0
					&& readMatrix(matrix, i - 1, j) + GAP
							== readMatrix(matrix, i, j)) {
				stack<int*> newSt = stack<int*>(temp[ii]);
				arr = (int*) malloc(2 * sizeof(int));
				arr[0] = i - 1;
				arr[1] = j;
				newSt.push(arr);
				stList.push_back(newSt);

				hadPath = true;
			}

			if (j - 1 >= 0
					&& readMatrix(matrix, i, j - 1) + GAP
							== readMatrix(matrix, i, j)) {
				stack<int*> newSt = stack<int*>(temp[ii]);
				arr = (int*) malloc(2 * sizeof(int));
				arr[0] = i;
				arr[1] = j - 1;
				newSt.push(arr);
				stList.push_back(newSt);

				while (newSt.size() > 0) {
					newSt.pop();
				}

				hadPath = true;
			}

			if (i - 1 >= 0 && j - 1 >= 0 && Y[i - 1] == X[j - 1]) {
				if (readMatrix(matrix, i - 1, j - 1) + MATCH
						== readMatrix(matrix, i, j)) {
					stack<int*> newSt = stack<int*>(temp[ii]);
					arr = (int*) malloc(2 * sizeof(int));
					arr[0] = i - 1;
					arr[1] = j - 1;
					newSt.push(arr);
					stList.push_back(newSt);
					hadPath = true;
				}

			} else if (i - 1 >= 0 && j - 1 >= 0 && Y[i - 1] != X[j - 1]) {
				if (readMatrix(matrix, i - 1, j - 1) + MISMATCH
						== readMatrix(matrix, i, j)) {
					stack<int*> newSt = stack<int*>(temp[ii]);
					arr = (int*) malloc(2 * sizeof(int));
					arr[0] = i - 1;
					arr[1] = j - 1;
					newSt.push(arr);
					stList.push_back(newSt);
					hadPath = true;
				}
			}
		}
		if (stList.size() == 0) {
			stList = temp;
		}
	}

	for (int ii = 0; ii < stList.size(); ++ii) {
		stack<int*> stack = stList[ii];
		vector<int*> path = vector<int*>();

		while (!stack.empty()) {
			int* arr = stack.top();
			path.push_back(arr);
			stack.pop();
		}

		string xSeq = "";
		string matchSeq = "";
		string ySeq = "";

		for (int k = 1; k < path.size(); k++) {
			int i0 = path[k - 1][0];
			int j0 = path[k - 1][1];
			int i1 = path[k][0];
			int j1 = path[k][1];

			// if a match move
			if (i1 == i0 + 1 && j1 == j0 + 1) {
				xSeq += X[j0];
				ySeq += Y[i0];

				matchSeq += X[j0] == Y[i0] ? "|" : " ";
			}
			// if X gap
			else if (i1 == i0 + 1 && j1 == j0) {
				xSeq += "-";
				ySeq += Y[i1];
				matchSeq += " ";
			}
			// if Y gap
			else if (i1 == i0 && j1 == j0 + 1) {
				xSeq += X[i1];
				ySeq += "-";
				matchSeq += " ";
			}

		}
		cout << "\nAlignment" << endl;
		cout << "\t" + xSeq << endl;
		cout << "\t" + matchSeq << endl;
		cout << "\t" + ySeq + "\n" << endl;
	}

}
